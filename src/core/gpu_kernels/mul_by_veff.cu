
#include <hip/hip_runtime.h>
// Copyright (c) 2013-2019 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file mul_by_veff.cu
 *
 *  \brief CUDA kernels to multiply wave-functions by effective potential.
 */

//#include "gpu/acc_common.hpp"
//#include "gpu/acc_runtime.hpp"
//
//template <typename T>
//__global__ void
//mul_by_veff_real_real_gpu_kernel(int nr__, T const* in__,T const* veff__, T* out__)
//{
//    int i = blockDim.x * blockIdx.x + threadIdx.x;
//    if (i < nr__) {
//        out__[i] = in__[i] * veff__[i];
//    }
//}
//
//template <typename T>
//__global__ void
//mul_by_veff_complex_real_gpu_kernel(int nr__, gpu_complex_type<T> const* in__, T const* veff__,
//        gpu_complex_type<T>* out__)
//{
//    int i = blockDim.x * blockIdx.x + threadIdx.x;
//    if (i < nr__) {
//        out__[i] = mul_accNumbers(veff__[i], in__[i]);
//    }
//}
//
//template <typename T>
//__global__ void
//mul_by_veff_complex_complex_gpu_kernel(int nr__, gpu_complex_type<T> const* in__, T pref__, T const* vx__,
//        T const* vy__, gpu_complex_type<T>* out__)
//{
//    int i = blockDim.x * blockIdx.x + threadIdx.x;
//    if (i < nr__) {
//        out__[i] = mul_accNumbers(in__[i], make_accComplex(vx__[i], pref__ * vy__[i]));
//    }
//}
//
//extern "C" {
//
//void
//mul_by_veff_real_real_gpu_float(int nr__, float const* in__, float const* veff__, float* out__)
//{
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(nr__, grid_t.x));
//
//    accLaunchKernel((mul_by_veff_real_real_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
//            nr__, in__, veff__, out__);
//}
//
//void
//mul_by_veff_real_real_gpu_double(int nr__, double const* in__, double const* veff__, double* out__)
//{
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(nr__, grid_t.x));
//
//    accLaunchKernel((mul_by_veff_real_real_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
//            nr__, in__, veff__, out__);
//}
//
//void
//mul_by_veff_complex_real_gpu_float(int nr__, gpu_complex_type<float> const* in__, float const* veff__,
//        gpu_complex_type<float>* out__)
//{
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(nr__, grid_t.x));
//
//    accLaunchKernel((mul_by_veff_complex_real_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
//            nr__, in__, veff__, out__);
//}
//
//void
//mul_by_veff_complex_real_gpu_double(int nr__, gpu_complex_type<double> const* in__, double const* veff__,
//        gpu_complex_type<double>* out__)
//{
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(nr__, grid_t.x));
//
//    accLaunchKernel((mul_by_veff_complex_real_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
//            nr__, in__, veff__, out__);
//}
//
//void
//mul_by_veff_complex_complex_gpu_float(int nr__, gpu_complex_type<float> const* in__, float pref__,
//    float const* vx__, float const* vy__, gpu_complex_type<float>* out__)
//{
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(nr__, grid_t.x));
//
//    accLaunchKernel((mul_by_veff_complex_complex_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
//            nr__, in__, pref__, vx__, vy__, out__);
//}
//
//void
//mul_by_veff_complex_complex_gpu_double(int nr__, gpu_complex_type<double> const* in__, double pref__,
//    double const* vx__, double const* vy__, gpu_complex_type<double>* out__)
//{
//    dim3 grid_t(64);
//    dim3 grid_b(num_blocks(nr__, grid_t.x));
//
//    accLaunchKernel((mul_by_veff_complex_complex_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
//            nr__, in__, pref__, vx__, vy__, out__);
//}

//}



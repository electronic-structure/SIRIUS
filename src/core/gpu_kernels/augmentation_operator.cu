#include "hip/hip_runtime.h"
/* This file is part of SIRIUS electronic structure library.
 *
 * Copyright (c), ETH Zurich.  All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

/** \file augmentation_operator.cu
 *
 *  \brief CUDA kernels to generate augmentation operator and its derivative.
 */

#include "core/acc/acc.hpp"
#include "core/acc/acc_common.hpp"
#include "core/acc/acc_runtime.hpp"

using namespace sirius;
using namespace sirius::acc;

__global__ void aug_op_pw_coeffs_gpu_kernel(int ngvec__, int const* gvec_shell__, int const* idx__, int idxmax__,
                                            acc_complex_double_t const* zilm__, int const* l_by_lm__, int lmmax__,
                                            double const* gc__, int ld0__, int ld1__,
                                            double const* gvec_rlm__, int ld2__,
                                            double const* ri_values__, int ld3__, int ld4__,
                                            double* q_pw__, int ld5__, double fourpi_omega__)

{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;
    int idx12 = blockIdx.y;
    int idxsh = gvec_shell__[igloc];

    if (igloc < ngvec__) {
        int lm1     = idx__[array2D_offset(0, idx12, 3)];
        int lm2     = idx__[array2D_offset(1, idx12, 3)];
        int idxrf12 = idx__[array2D_offset(2, idx12, 3)];

        acc_complex_double_t z = make_accDoubleComplex(0, 0);
        for (int lm = 0; lm < lmmax__; lm++) {
            double d = gvec_rlm__[array2D_offset(lm, igloc, ld2__)] *
                ri_values__[array3D_offset(idxrf12, l_by_lm__[lm], idxsh, ld3__, ld4__)] *
                gc__[array3D_offset(lm, lm2, lm1, ld0__, ld1__)];
            z.x += d * zilm__[lm].x;
            z.y -= d * zilm__[lm].y;
        }
        q_pw__[array2D_offset(idx12, 2 * igloc,     ld5__)] = z.x * fourpi_omega__;
        q_pw__[array2D_offset(idx12, 2 * igloc + 1, ld5__)] = z.y * fourpi_omega__;
    }
}

extern "C" void aug_op_pw_coeffs_gpu(int ngvec__, int const* gvec_shell__, int const* idx__, int idxmax__,
                                     acc_complex_double_t const* zilm__, int const* l_by_lm__, int lmmax__,
                                     double const* gc__, int ld0__, int ld1__,
                                     double const* gvec_rlm__, int ld2__,
                                     double const* ri_values__, int ld3__, int ld4__,
                                     double* q_pw__, int ld5__, double fourpi_omega__)
{
    dim3 grid_t(32);
    dim3 grid_b(num_blocks(ngvec__, grid_t.x), idxmax__);

    accLaunchKernel((aug_op_pw_coeffs_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0,
        ngvec__, gvec_shell__, idx__, idxmax__, zilm__, l_by_lm__, lmmax__, gc__, ld0__, ld1__, gvec_rlm__, ld2__,
        ri_values__, ld3__, ld4__, q_pw__, ld5__, fourpi_omega__);
}

__global__ void aug_op_pw_coeffs_deriv_gpu_kernel(int ngvec__, int const* gvec_shell__, double const* gvec_cart__,
                                                  int const* idx__, int idxmax__,
                                                  double const* gc__, int ld0__, int ld1__,
                                                  double const* rlm__, double const* rlm_dg__, int ld2__,
                                                  double const* ri_values__, double const* ri_dg_values__, int ld3__,
                                                  int ld4__, double* q_pw__, int ld5__, double fourpi__, int nu__,
                                                  int lmax_q__)

{
    int igloc = blockDim.x * blockIdx.x + threadIdx.x;
    int idx12 = blockIdx.y;
    int idxsh = gvec_shell__[igloc];

    if (igloc < ngvec__) {
        int lm1     = idx__[array2D_offset(0, idx12, 3)];
        int lm2     = idx__[array2D_offset(1, idx12, 3)];
        int idxrf12 = idx__[array2D_offset(2, idx12, 3)];
        double gvc_nu = gvec_cart__[array2D_offset(nu__, igloc, 3)];

        acc_complex_double_t z = make_accDoubleComplex(0, 0);
        acc_complex_double_t phase = make_accDoubleComplex(1, 0);
        int lm = 0;
        for (int l = 0; l <= lmax_q__; l++) {
            double d1 = 0;
            double d2 = 0;
            for (int m = -l; m <= l; m++, lm++) {
                double gc = gc__[array3D_offset(lm, lm2, lm1, ld0__, ld1__)];
                d1 += rlm_dg__[array3D_offset(lm, nu__, igloc, ld2__, 3)] * gc;
                d2 += rlm__[array2D_offset(lm, igloc, ld2__)] * gc;
            }
            double d = d1 * ri_values__[array3D_offset(l, idxrf12, idxsh, ld3__, ld4__)] +
                       d2 * ri_dg_values__[array3D_offset(l, idxrf12, idxsh, ld3__, ld4__)] * gvc_nu;
            z.x += d * phase.x;
            z.y -= d * phase.y;
            /* i^l */
            phase = accCmul(phase, make_accDoubleComplex(0, 1));
        }
        q_pw__[array2D_offset(idx12, 2 * igloc,     ld5__)] = z.x * fourpi__;
        q_pw__[array2D_offset(idx12, 2 * igloc + 1, ld5__)] = z.y * fourpi__;
    }
}

extern "C" void aug_op_pw_coeffs_deriv_gpu(int ngvec__, int const* gvec_shell__, double const* gvec_cart__,
                                           int const* idx__, int idxmax__,
                                           double const* gc__, int ld0__, int ld1__,
                                           double const* rlm__, double const* rlm_dg__, int ld2__,
                                           double const* ri_values__, double const* ri_dg_values__, int ld3__, int ld4__,
                                           double* q_pw__, int ld5__, double fourpi__, int nu__, int lmax_q__)
{
    dim3 grid_t(32);
    dim3 grid_b(num_blocks(ngvec__, grid_t.x), idxmax__);

    accLaunchKernel((aug_op_pw_coeffs_deriv_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0,
        ngvec__, gvec_shell__, gvec_cart__, idx__, idxmax__, gc__, ld0__, ld1__,
        rlm__, rlm_dg__, ld2__, ri_values__, ri_dg_values__, ld3__, ld4__, q_pw__, ld5__, fourpi__, nu__, lmax_q__);
}


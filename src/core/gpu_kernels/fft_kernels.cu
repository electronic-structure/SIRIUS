#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file fft_kernels.cu
 *
 *  \brief Contains implementation of CUDA and ROCM kernels necessary for a FFT driver.
 */

#include <stdio.h>
#include "acc_common.hpp"
#include "acc_runtime.hpp"

//NOTE: HIP will call the corresponding CUDA function if compiled with CUDA support


template <int direction>
__global__ void repack_z_buffer_gpu_kernel(int size_z,
                                           int num_zcol_loc,
                                           int const* local_z_offsets,
                                           int const* local_z_sizes,
                                           acc_complex_double_t* z_sticks_local,
                                           acc_complex_double_t* a2a_buffer)
{
    int iz = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
    int izcol = hipBlockIdx_y;
    int rank = hipBlockIdx_z;

    int local_zsize = local_z_sizes[rank];
    if (iz < local_zsize) {
        int offs = local_z_offsets[rank];
        if (direction == -1) {
            z_sticks_local[offs + iz + izcol * size_z] = a2a_buffer[offs * num_zcol_loc + izcol * local_zsize + iz];
        }
        if (direction == 1) {
            a2a_buffer[offs * num_zcol_loc + izcol * local_zsize + iz] = z_sticks_local[offs + iz + izcol * size_z];
        }
    }
}

extern "C" void repack_z_buffer_gpu(int direction,
                                    int num_ranks,
                                    int size_z,
                                    int num_zcol_loc,
                                    int zcol_max_size,
                                    int const* local_z_offsets,
                                    int const* local_z_sizes,
                                    acc_complex_double_t* z_sticks_local,
                                    acc_complex_double_t* a2a_buffer)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(zcol_max_size, grid_t.x), num_zcol_loc, num_ranks);

    if (direction == 1) {
        accLaunchKernel((repack_z_buffer_gpu_kernel<1>), dim3(grid_b), dim3(grid_t), 0, 0, 
            size_z,
            num_zcol_loc,
            local_z_offsets,
            local_z_sizes,
            z_sticks_local,
            a2a_buffer
        );
    } else {
        accLaunchKernel((repack_z_buffer_gpu_kernel<-1>), dim3(grid_b), dim3(grid_t), 0, 0, 
            size_z,
            num_zcol_loc,
            local_z_offsets,
            local_z_sizes,
            z_sticks_local,
            a2a_buffer
        );
    }
}



__global__ void batch_load_gpu_kernel(int                    fft_size, 
                                      int                    num_pw_components, 
                                      int const*             map, 
                                      acc_complex_double_t const* data, 
                                      acc_complex_double_t*       fft_buffer)
{
    int i = hipBlockIdx_y;
    int idx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    if (idx < num_pw_components) {
        fft_buffer[array2D_offset(map[idx], i, fft_size)] = data[array2D_offset(idx, i, num_pw_components)];
    }
}

extern "C" void batch_load_gpu(int                    fft_size,
                               int                    num_pw_components, 
                               int                    num_fft,
                               int const*             map, 
                               acc_complex_double_t const* data, 
                               acc_complex_double_t*       fft_buffer,
                               int                    stream_id__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_pw_components, grid_t.x), num_fft);

    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    acc::zero(fft_buffer, fft_size*num_fft, stream_id(stream_id__));

    accLaunchKernel((batch_load_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, stream, 
        fft_size,
        num_pw_components,
        map,
        data, 
        fft_buffer
    );
}

__global__ void batch_unload_gpu_kernel(int                    fft_size, 
                                        int                    num_pw_components, 
                                        int const*             map, 
                                        acc_complex_double_t const* fft_buffer,
                                        acc_complex_double_t*       data,
                                        double                 alpha,
                                        double                 beta)
{
    int i = hipBlockIdx_y;
    int idx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    if (idx < num_pw_components) {
        acc_complex_double_t z1 = data[array2D_offset(idx, i, num_pw_components)];
        acc_complex_double_t z2 = fft_buffer[array2D_offset(map[idx], i, fft_size)];
        data[array2D_offset(idx, i, num_pw_components)] = make_accDoubleComplex(alpha * z1.x + beta * z2.x, alpha * z1.y + beta * z2.y);

        //data[array2D_offset(idx, i, num_pw_components)] = hipCadd(
        //    hipCmul(make_hipDoubleComplex(alpha, 0), data[array2D_offset(idx, i, num_pw_components)]),
        //    hipCmul(make_hipDoubleComplex(beta, 0), fft_buffer[array2D_offset(map[idx], i, fft_size)]));
    }
}

/// Unload data from FFT buffer.
/** The following operation is executed:
 *  data[ig] = alpha * data[ig] + beta * fft_buffer[map[ig]] */
extern "C" void batch_unload_gpu(int                    fft_size,
                                 int                    num_pw_components,
                                 int                    num_fft,
                                 int const*             map, 
                                 acc_complex_double_t const* fft_buffer, 
                                 acc_complex_double_t*       data,
                                 double                 alpha,
                                 double                 beta,
                                 int                    stream_id__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_pw_components, grid_t.x), num_fft);

    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    if (alpha == 0) {
        acc::zero(data, num_pw_components, stream_id(stream_id__));
    }

    accLaunchKernel((batch_unload_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, stream, 
        fft_size, 
        num_pw_components, 
        map, 
        fft_buffer,
        data,
        alpha,
        beta
    );
}

__global__ void load_x0y0_col_gpu_kernel(int                    z_col_size,
                                         int const*             map,
                                         acc_complex_double_t const* data,
                                         acc_complex_double_t*       fft_buffer)

{
    int idx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    if (idx < z_col_size) {
        fft_buffer[map[idx]] = make_accDoubleComplex(data[idx].x, -data[idx].y);
    }
}

extern "C" void load_x0y0_col_gpu(int                    z_col_size,
                                  int const*             map,
                                  acc_complex_double_t const* data,
                                  acc_complex_double_t*       fft_buffer,
                                  int                    stream_id__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(z_col_size, grid_t.x));

    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    accLaunchKernel((load_x0y0_col_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, stream, 
        z_col_size,
        map,
        data,
        fft_buffer
    );
}

template <int direction, bool conjugate>
__global__ void pack_unpack_z_cols_gpu_kernel(acc_complex_double_t* z_cols_packed__,
                                              acc_complex_double_t* fft_buf__,
                                              int              size_x__,
                                              int              size_y__,
                                              int              size_z__,
                                              int              num_z_cols__,
                                              int const*       z_col_pos__)
{
    int icol = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    int iz = hipBlockIdx_y;
    int size_xy = size_x__ * size_y__;
    if (icol < num_z_cols__) {
        int ipos = z_col_pos__[icol];
        /* load into buffer */
        if (direction == 1) {
            if (conjugate) {
                fft_buf__[array2D_offset(ipos, iz, size_xy)].x = z_cols_packed__[array2D_offset(iz, icol, size_z__)].x;
                fft_buf__[array2D_offset(ipos, iz, size_xy)].y = -z_cols_packed__[array2D_offset(iz, icol, size_z__)].y;
            }
            else {
                fft_buf__[array2D_offset(ipos, iz, size_xy)] = z_cols_packed__[array2D_offset(iz, icol, size_z__)];
            }
        }
        if (direction == -1) {
            z_cols_packed__[array2D_offset(iz, icol, size_z__)] = fft_buf__[array2D_offset(ipos, iz, size_xy)];
        }
    }
}

extern "C" void unpack_z_cols_gpu(acc_complex_double_t* z_cols_packed__,
                                  acc_complex_double_t* fft_buf__,
                                  int              size_x__,
                                  int              size_y__,
                                  int              size_z__,
                                  int              num_z_cols__,
                                  int const*       z_col_pos__,
                                  bool             use_reduction__, 
                                  int              stream_id__)
{
    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_z_cols__, grid_t.x), size_z__);

    acc::zero(fft_buf__, size_x__ * size_y__ * size_z__, stream_id(stream_id__));

    accLaunchKernel((pack_unpack_z_cols_gpu_kernel<1, false>), dim3(grid_b), dim3(grid_t), 0, stream, 
        z_cols_packed__,
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__,
        z_col_pos__
    );
    if (use_reduction__) {
        accLaunchKernel((pack_unpack_z_cols_gpu_kernel<1, true>), dim3(grid_b), dim3(grid_t), 0, stream, 
            &z_cols_packed__[size_z__], // skip first column for {-x, -y} coordinates
            fft_buf__,
            size_x__,
            size_y__,
            size_z__,
            num_z_cols__ - 1,
            &z_col_pos__[num_z_cols__ + 1] // skip first column for {-x, -y} coordinates
        );
    }
}

extern "C" void pack_z_cols_gpu(acc_complex_double_t* z_cols_packed__,
                                acc_complex_double_t* fft_buf__,
                                int              size_x__,
                                int              size_y__,
                                int              size_z__,
                                int              num_z_cols__,
                                int const*       z_col_pos__,
                                int              stream_id__)
{
    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_z_cols__, grid_t.x), size_z__);

    accLaunchKernel((pack_unpack_z_cols_gpu_kernel<-1, false>), dim3(grid_b), dim3(grid_t), 0, stream, 
        z_cols_packed__,
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__,
        z_col_pos__
    );
}

template <int direction, bool conjugate>
__global__ void pack_unpack_two_z_cols_gpu_kernel(acc_complex_double_t* z_cols_packed1__,
                                                  acc_complex_double_t* z_cols_packed2__,
                                                  acc_complex_double_t* fft_buf__,
                                                  int              size_x__,
                                                  int              size_y__,
                                                  int              size_z__,
                                                  int              num_z_cols__,
                                                  int const*       z_col_pos__)
{
    int icol = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    int iz = hipBlockIdx_y;
    int size_xy = size_x__ * size_y__;
    if (icol < num_z_cols__) {
        /* load into buffer */
        if (direction == 1) {
            int ipos = z_col_pos__[icol];
            acc_complex_double_t z1 = z_cols_packed1__[array2D_offset(iz, icol, size_z__)];
            acc_complex_double_t z2 = z_cols_packed2__[array2D_offset(iz, icol, size_z__)];
            if (conjugate) {
                /* conj(z1) + I * conj(z2) */
                fft_buf__[array2D_offset(ipos, iz, size_xy)] = make_accDoubleComplex(z1.x + z2.y, z2.x - z1.y);
            }
            else {
                /* z1 + I * z2 */
                fft_buf__[array2D_offset(ipos, iz, size_xy)] = make_accDoubleComplex(z1.x - z2.y, z1.y + z2.x);
            }
        }
        if (direction == -1) {
            int ipos1 = z_col_pos__[icol];
            int ipos2 = z_col_pos__[num_z_cols__ + icol];
            acc_complex_double_t z1 = fft_buf__[array2D_offset(ipos1, iz, size_xy)];
            acc_complex_double_t z2 = fft_buf__[array2D_offset(ipos2, iz, size_xy)];

            z_cols_packed1__[array2D_offset(iz, icol, size_z__)] = make_accDoubleComplex(0.5 * (z1.x + z2.x), 0.5 * (z1.y - z2.y));
            z_cols_packed2__[array2D_offset(iz, icol, size_z__)] = make_accDoubleComplex(0.5 * (z1.y + z2.y), 0.5 * (z2.x - z1.x));
        }
    }
}

extern "C" void unpack_z_cols_2_gpu(acc_complex_double_t* z_cols_packed1__,
                                    acc_complex_double_t* z_cols_packed2__,
                                    acc_complex_double_t* fft_buf__,
                                    int              size_x__,
                                    int              size_y__,
                                    int              size_z__,
                                    int              num_z_cols__,
                                    int const*       z_col_pos__,
                                    int              stream_id__)
{
    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_z_cols__, grid_t.x), size_z__);

    acc::zero(fft_buf__, size_x__ * size_y__ * size_z__, stream_id(stream_id__));

    accLaunchKernel((pack_unpack_two_z_cols_gpu_kernel<1, false>), dim3(grid_b), dim3(grid_t), 0, stream, 
        z_cols_packed1__,
        z_cols_packed2__,
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__,
        z_col_pos__
    );
    accLaunchKernel((pack_unpack_two_z_cols_gpu_kernel<1, true>), dim3(grid_b), dim3(grid_t), 0, stream, 
        &z_cols_packed1__[size_z__], // skip first column for {-x, -y} coordinates
        &z_cols_packed2__[size_z__], // skip first column for {-x, -y} coordinates
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__ - 1,
        &z_col_pos__[num_z_cols__ + 1] // skip first column for {-x, -y} coordinates
    );
}

extern "C" void pack_z_cols_2_gpu(acc_complex_double_t* z_cols_packed1__,
                                  acc_complex_double_t* z_cols_packed2__,
                                  acc_complex_double_t* fft_buf__,
                                  int              size_x__,
                                  int              size_y__,
                                  int              size_z__,
                                  int              num_z_cols__,
                                  int const*       z_col_pos__,
                                  int              stream_id__)
{
    acc_stream_t stream = (acc_stream_t) acc::stream(stream_id(stream_id__));

    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_z_cols__, grid_t.x), size_z__);

    accLaunchKernel((pack_unpack_two_z_cols_gpu_kernel<-1, false>), dim3(grid_b), dim3(grid_t), 0, stream, 
        z_cols_packed1__,
        z_cols_packed2__,
        fft_buf__,
        size_x__,
        size_y__,
        size_z__,
        num_z_cols__,
        z_col_pos__
    );
}


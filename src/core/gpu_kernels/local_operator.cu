#include "hip/hip_runtime.h"
// Copyright (c) 2013-2022 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file local_operator.cu
 *
 *  \brief GPU kernels and API for application of the local operator.
 */

#include "core/acc/acc_common.hpp"
#include "core/acc/acc_runtime.hpp"

using namespace sirius;
using namespace sirius::acc;

template <typename T>
__global__ void
add_to_hphi_pw_gpu_kernel(int num_gvec__, gpu_complex_type<T> const* vphi__, gpu_complex_type<T>* hphi__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        hphi__[ig] = add_accNumbers(hphi__[ig], vphi__[ig]);
    }
}

template <typename T>
__global__ void
add_to_hphi_pw_gpu_kernel(int num_gvec__, T const* pw_ekin__, gpu_complex_type<T> const* phi__,
        gpu_complex_type<T> const* vphi__, gpu_complex_type<T>* hphi__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        auto z1    = add_accNumbers(vphi__[ig], mul_accNumbers(pw_ekin__[ig], phi__[ig]));
        hphi__[ig] = add_accNumbers(hphi__[ig], z1);
    }
}

template <typename T>
__global__ void
add_to_hphi_lapw_gpu_kernel(int num_gvec__, gpu_complex_type<T>* const p__, T const* gkvec_cart__,
        gpu_complex_type<T>* hphi__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        /* hphi[ig] = hphi[ig] + 1/2 p_{x,y,z}[ig] * G_{x,y,z}[ig] */
        hphi__[ig] = add_accNumbers(hphi__[ig], mul_accNumbers(0.5 * gkvec_cart__[ig], p__[ig]));
    }
}

template <typename T>
__global__ void
grad_phi_lapw_gpu_kernel(int num_gvec__, gpu_complex_type<T>* const phi__, T const* gkvec_cart__,
        gpu_complex_type<T>* p__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        p__[ig] = mul_accNumbers(gkvec_cart__[ig], phi__[ig]);
    }
}

template <typename T>
__global__ void
mul_by_veff_real_real_gpu_kernel(int nr__, T const* in__,T const* veff__, T* out__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        out__[i] = in__[i] * veff__[i];
    }
}

template <typename T>
__global__ void
mul_by_veff_complex_real_gpu_kernel(int nr__, gpu_complex_type<T> const* in__, T const* veff__,
        gpu_complex_type<T>* out__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        out__[i] = mul_accNumbers(veff__[i], in__[i]);
    }
}

template <typename T>
__global__ void
mul_by_veff_complex_complex_gpu_kernel(int nr__, gpu_complex_type<T> const* in__, T pref__, T const* vx__,
        T const* vy__, gpu_complex_type<T>* out__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        out__[i] = mul_accNumbers(in__[i], make_accComplex(vx__[i], pref__ * vy__[i]));
    }
}

/// Update the hphi wave functions.
/** The following operation is performed:
 *    hphi[ig] += (alpha *  pw_ekin[ig] * phi[ig] + vphi[ig])
 */
extern "C" {

void
add_to_hphi_pw_gpu_float(int num_gvec__, int add_ekin__, float const* pw_ekin__, gpu_complex_type<float> const* phi__,
    gpu_complex_type<float> const* vphi__, gpu_complex_type<float>* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    if (add_ekin__) {
        accLaunchKernel((add_to_hphi_pw_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
                num_gvec__, pw_ekin__, phi__, vphi__, hphi__);
    } else {
        accLaunchKernel((add_to_hphi_pw_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
                num_gvec__, vphi__, hphi__);
    }
}

void
add_to_hphi_pw_gpu_double(int num_gvec__, int add_ekin__, double const* pw_ekin__, gpu_complex_type<double> const* phi__,
    gpu_complex_type<double> const* vphi__, gpu_complex_type<double>* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    if (add_ekin__) {
        accLaunchKernel((add_to_hphi_pw_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
                num_gvec__, pw_ekin__, phi__, vphi__, hphi__);
    } else {
        accLaunchKernel((add_to_hphi_pw_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
                num_gvec__, vphi__, hphi__);
    }
}

void
add_to_hphi_lapw_gpu_float(int num_gvec__, gpu_complex_type<float>* const p__, float const* gkvec_cart__,
        gpu_complex_type<float>* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    accLaunchKernel((add_to_hphi_lapw_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
            num_gvec__, p__, gkvec_cart__, hphi__);
}

void
grad_phi_lapw_gpu_float(int num_gvec__, gpu_complex_type<float>* const p__, float const* gkvec_cart__,
        gpu_complex_type<float>* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    accLaunchKernel((grad_phi_lapw_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
            num_gvec__, p__, gkvec_cart__, hphi__);
}

void
add_to_hphi_lapw_gpu_double(int num_gvec__, gpu_complex_type<double>* const p__, double const* gkvec_cart__,
        gpu_complex_type<double>* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    accLaunchKernel((add_to_hphi_lapw_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
            num_gvec__, p__, gkvec_cart__, hphi__);
}

void
grad_phi_lapw_gpu_double(int num_gvec__, gpu_complex_type<double>* const p__, double const* gkvec_cart__,
        gpu_complex_type<double>* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    accLaunchKernel((grad_phi_lapw_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
            num_gvec__, p__, gkvec_cart__, hphi__);
}

void
mul_by_veff_real_real_gpu_float(int nr__, float const* in__, float const* veff__, float* out__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_real_real_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
            nr__, in__, veff__, out__);
}

void
mul_by_veff_real_real_gpu_double(int nr__, double const* in__, double const* veff__, double* out__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_real_real_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
            nr__, in__, veff__, out__);
}

void
mul_by_veff_complex_real_gpu_float(int nr__, gpu_complex_type<float> const* in__, float const* veff__,
        gpu_complex_type<float>* out__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_real_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
            nr__, in__, veff__, out__);
}

void
mul_by_veff_complex_real_gpu_double(int nr__, gpu_complex_type<double> const* in__, double const* veff__,
        gpu_complex_type<double>* out__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_real_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
            nr__, in__, veff__, out__);
}

void
mul_by_veff_complex_complex_gpu_float(int nr__, gpu_complex_type<float> const* in__, float pref__,
    float const* vx__, float const* vy__, gpu_complex_type<float>* out__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_complex_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0,
            nr__, in__, pref__, vx__, vy__, out__);
}

void
mul_by_veff_complex_complex_gpu_double(int nr__, gpu_complex_type<double> const* in__, double pref__,
    double const* vx__, double const* vy__, gpu_complex_type<double>* out__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_complex_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0,
            nr__, in__, pref__, vx__, vy__, out__);
}

}

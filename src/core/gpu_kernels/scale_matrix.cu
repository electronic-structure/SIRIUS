#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file scale_matrix.cu
 *
 *  \brief Contains implementation of CUDA kernels to scale matrix elements (rows or columns).
 */

#include "core/acc/acc_common.hpp"
#include "core/acc/acc_runtime.hpp"

using namespace sirius;
using namespace sirius::acc;

template <typename T>
__global__ void scale_matrix_columns_gpu_kernel(int nrow, gpu_complex_type<T>* mtrx, T* a);

template <>
__global__ void scale_matrix_columns_gpu_kernel<double>
(
    int nrow,
    acc_complex_double_t* mtrx,
    double* a
)
{
    int icol = blockIdx.y;
    int irow = blockIdx.x * blockDim.x + threadIdx.x;
    if (irow < nrow) {
        mtrx[array2D_offset(irow, icol, nrow)] =
            accCmul(mtrx[array2D_offset(irow, icol, nrow)], make_accDoubleComplex(a[icol], 0));
    }
}

template <>
__global__ void scale_matrix_columns_gpu_kernel<float>
    (
        int nrow,
        acc_complex_float_t* mtrx,
        float* a
    )
{
    int icol = blockIdx.y;
    int irow = blockIdx.x * blockDim.x + threadIdx.x;
    if (irow < nrow) {
        mtrx[array2D_offset(irow, icol, nrow)] =
            accCmulf(mtrx[array2D_offset(irow, icol, nrow)], make_accFloatComplex(a[icol], 0));
    }
}

// scale each column of the matrix by a column-dependent constant
extern "C" void scale_matrix_columns_gpu_double(int nrow,
                                                 int ncol,
                                                 acc_complex_double_t* mtrx,
                                                 double* a)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow, grid_t.x), ncol);

    accLaunchKernel((scale_matrix_columns_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0, nrow, mtrx, a);
}

extern "C" void scale_matrix_columns_gpu_float(int nrow,
                                                int ncol,
                                                acc_complex_float_t* mtrx,
                                                float* a)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow, grid_t.x), ncol);

    accLaunchKernel((scale_matrix_columns_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0, nrow, mtrx, a);
}

__global__ void scale_matrix_rows_gpu_kernel
(
    int nrow__,
    acc_complex_double_t* mtrx__,
    double const* v__
)
{
    int icol = blockIdx.y;
    int irow = blockDim.x * blockIdx.x + threadIdx.x;
    if (irow < nrow__) {
        acc_complex_double_t z = mtrx__[array2D_offset(irow, icol, nrow__)];
        mtrx__[array2D_offset(irow, icol, nrow__)] = make_accDoubleComplex(z.x * v__[irow], z.y * v__[irow]);
    }
}

// scale each row of the matrix by a row-dependent constant
extern "C" void scale_matrix_rows_gpu(int nrow__,
                                      int ncol__,
                                      acc_complex_double_t* mtrx__,
                                      double const* v__)
{
    dim3 grid_t(256);
    dim3 grid_b(num_blocks(nrow__, grid_t.x), ncol__);

    accLaunchKernel((scale_matrix_rows_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        nrow__,
        mtrx__,
        v__
    );
}

__global__ void scale_matrix_elements_gpu_kernel
(
    acc_complex_double_t* mtrx__,
    int ld__,
    int nrow__,
    double beta__
)
{
    int icol = blockIdx.y;
    int irow = blockDim.x * blockIdx.x + threadIdx.x;
    if (irow < nrow__) {
        acc_complex_double_t z = mtrx__[array2D_offset(irow, icol, ld__)];
        mtrx__[array2D_offset(irow, icol, ld__)] = make_accDoubleComplex(z.x * beta__, z.y * beta__);
    }
}

extern "C" void scale_matrix_elements_gpu(acc_complex_double_t* ptr__,
                                          int ld__,
                                          int nrow__,
                                          int ncol__,
                                          double beta__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nrow__, grid_t.x), ncol__);

    accLaunchKernel((scale_matrix_elements_gpu_kernel), dim3(grid_b), dim3(grid_t), 0, 0, 
        ptr__,
        ld__,
        nrow__,
        beta__
    );
}

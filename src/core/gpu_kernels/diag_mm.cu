#include "hip/hip_runtime.h"
// Copyright (c) 2013-2023 Simon Pintarelli
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file diag_mm.cu
 *
 *  \brief
 */

#include "diag_mm.hpp"
#include "acc_runtime.hpp"
#include "acc.hpp"

template <class T>
__global__ std::enable_if_t<!std::is_same<acc_complex_double_t, T>::value>
diag_mm(const T* diag, int n, const T* X, int lda_x, int ncols, T* Y, int lda_y, T alpha)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < ncols && row < n) {
        T X_elem                 = *(X + lda_x * col + row);
        T D                      = diag[row];
        *(Y + lda_y * col + row) = alpha * D * X_elem;
    }
}

template <class T>
__global__ std::enable_if_t<std::is_same<acc_complex_double_t, T>::value>
diag_mm(const T* diag, int n, const T* X, int lda_x, int ncols, T* Y, int lda_y, T alpha)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < ncols && row < n) {
        acc_complex_double_t X_elem = *(X + lda_x * col + row);
        acc_complex_double_t D      = diag[row];
        *(Y + lda_y * col + row)    = accCmul(accCmul(alpha, D), X_elem);
    }
}

template <class T>
void
call_diagmm(const T* diag, int n, const T* X, int lda_x, int ncols, T* Y, int lda_y, T alpha)
{
    int numthreads = 32;
    dim3 threadsPerBlock(numthreads, numthreads);

    int num_block_rows = (n + threadsPerBlock.x - 1) / threadsPerBlock.x;
    int num_block_cols = (ncols + threadsPerBlock.y - 1) / threadsPerBlock.y;
    dim3 numBlocks(num_block_rows, num_block_cols);

    diag_mm<<<numBlocks, threadsPerBlock>>>(diag, n, X, lda_x, ncols, Y, lda_y, alpha);
}

extern "C" {
void
ddiagmm(const double* diag, int n, const double* X, int lda_x, int ncols, double* Y, int lda_y, double alpha)
{
    call_diagmm(diag, n, X, lda_x, ncols, Y, lda_y, alpha);
}

void
sdiagmm(const float* diag, int n, const float* X, int lda_x, int ncols, float* Y, int lda_y, float alpha)
{
    call_diagmm(diag, n, X, lda_x, ncols, Y, lda_y, alpha);
}
void
zdiagmm(const std::complex<double>* diag, int n, const std::complex<double>* X, int lda_x, int ncols,
        std::complex<double>* Y, int lda_y, std::complex<double> alpha)
{
    call_diagmm(reinterpret_cast<const acc_complex_double_t*>(diag), n,
                reinterpret_cast<const acc_complex_double_t*>(X), lda_x, ncols,
                reinterpret_cast<acc_complex_double_t*>(Y), lda_y, acc_complex_double_t{alpha.real(), alpha.imag()});
}
}

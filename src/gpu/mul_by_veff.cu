#include "hip/hip_runtime.h"
// Copyright (c) 2013-2019 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file mul_by_veff.cu
 *
 *  \brief CUDA kernels to multiply wave-functions by effective potential.
 */

#include "gpu/acc_common.hpp"
#include "gpu/acc_runtime.hpp"

template <typename T>
__global__ void mul_by_veff_real_real_gpu_kernel(int nr__, T* buf__, T const* veff__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        buf__[i] *= veff__[i];
    }
}

extern "C" void mul_by_veff_real_real_gpu_float(int nr__, float* buf__, float const* veff__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_real_real_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0, nr__, buf__, veff__);
}

extern "C" void mul_by_veff_real_real_gpu_double(int nr__, double* buf__, double const* veff__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_real_real_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0, nr__, buf__, veff__);
}

template <typename T>
__global__ void mul_by_veff_complex_real_gpu_kernel(int nr__, gpu_complex_type<T>* buf__, T const* veff__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        buf__[i].x *= veff__[i];
        buf__[i].y *= veff__[i];
    }
}

extern "C" void mul_by_veff_complex_real_gpu_float(int nr__, acc_complex_float_t* buf__, float const* veff__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_real_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0, nr__, buf__, veff__);
}

extern "C" void mul_by_veff_complex_real_gpu_double(int nr__, acc_complex_double_t* buf__, double const* veff__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_real_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0, nr__, buf__, veff__);
}

template <typename T>
__global__ void mul_by_veff_complex_complex_gpu_kernel(int nr__, gpu_complex_type<T>* buf__, T pref__,
                                                       T const* vx__, T const* vy__);

template <>
__global__ void mul_by_veff_complex_complex_gpu_kernel<float>(int nr__, acc_complex_float_t* buf__, float pref__,
                                                               float const* vx__, float const* vy__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        acc_complex_float_t z = buf__[i];
        acc_complex_float_t v = make_accFloatComplex(vx__[i], pref__ * vy__[i]);
        buf__[i] = accCmulf(z, v);
    }
}

template <>
__global__ void mul_by_veff_complex_complex_gpu_kernel<double>(int nr__, acc_complex_double_t* buf__, double pref__,
                                                               double const* vx__, double const* vy__)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nr__) {
        acc_complex_double_t z = buf__[i];
        acc_complex_double_t v = make_accDoubleComplex(vx__[i], pref__ * vy__[i]);
        buf__[i] = accCmul(z, v);
    }
}

extern "C" void mul_by_veff_complex_complex_gpu_float(int nr__, acc_complex_float_t* buf__, float pref__,
                                                      float const* vx__, float const* vy__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_complex_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0, nr__, buf__,
                    pref__, vx__, vy__);
}

extern "C" void mul_by_veff_complex_complex_gpu_double(int nr__, acc_complex_double_t* buf__, double pref__,
                                                       double const* vx__, double const* vy__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(nr__, grid_t.x));

    accLaunchKernel((mul_by_veff_complex_complex_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0, nr__, buf__,
                    pref__, vx__, vy__);
}

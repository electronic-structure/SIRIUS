#include "hip/hip_runtime.h"
// Copyright (c) 2013-2018 Anton Kozhevnikov, Thomas Schulthess
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification, are permitted provided that
// the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this list of conditions and the
//    following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions
//    and the following disclaimer in the documentation and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

/** \file add_pw_ekin.cu
 *
 *  \brief CUDA kernel for the hphi update.
 */

#include "gpu/acc_common.hpp"
#include "gpu/acc_runtime.hpp"

template <typename T>
__global__ void add_pw_ekin_gpu_kernel(int num_gvec__,
                                       T alpha__,
                                       T const* pw_ekin__,
                                       gpu_complex_type<T> const* phi__,
                                       gpu_complex_type<T> const* vphi__,
                                       gpu_complex_type<T>* hphi__);

template <>
__global__ void add_pw_ekin_gpu_kernel<float>(int num_gvec__,
                                              float alpha__,
                                              float const* pw_ekin__,
                                              acc_complex_float_t const* phi__,
                                              acc_complex_float_t const* vphi__,
                                              acc_complex_float_t* hphi__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        acc_complex_float_t z1 = accCaddf(vphi__[ig], make_accFloatComplex(alpha__ * pw_ekin__[ig] * phi__[ig].x,
                                                                           alpha__ * pw_ekin__[ig] * phi__[ig].y));
        hphi__[ig]             = accCaddf(hphi__[ig], z1);
    }
}

template <>
__global__ void add_pw_ekin_gpu_kernel<double>(int num_gvec__,
                                               double alpha__,
                                               double const* pw_ekin__,
                                               acc_complex_double_t const* phi__,
                                               acc_complex_double_t const* vphi__,
                                               acc_complex_double_t* hphi__)
{
    int ig = blockIdx.x * blockDim.x + threadIdx.x;
    if (ig < num_gvec__) {
        acc_complex_double_t z1 = accCadd(vphi__[ig], make_accDoubleComplex(alpha__ * pw_ekin__[ig] * phi__[ig].x,
                                                                            alpha__ * pw_ekin__[ig] * phi__[ig].y));
        hphi__[ig]              = accCadd(hphi__[ig], z1);
    }
}

/// Update the hphi wave functions.
/** The following operation is performed:
 *    hphi[ig] += (alpha *  pw_ekin[ig] * phi[ig] + vphi[ig])
 */
extern "C" void add_pw_ekin_gpu_float(int num_gvec__,
                                       float alpha__,
                                       float const* pw_ekin__,
                                       acc_complex_float_t const* phi__,
                                       acc_complex_float_t const* vphi__,
                                       acc_complex_float_t* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    accLaunchKernel((add_pw_ekin_gpu_kernel<float>), dim3(grid_b), dim3(grid_t), 0, 0, num_gvec__, alpha__, pw_ekin__,
                    phi__, vphi__, hphi__);
}

extern "C" void add_pw_ekin_gpu_double(int num_gvec__,
                                double alpha__,
                                double const* pw_ekin__,
                                acc_complex_double_t const* phi__,
                                acc_complex_double_t const* vphi__,
                                acc_complex_double_t* hphi__)
{
    dim3 grid_t(64);
    dim3 grid_b(num_blocks(num_gvec__, grid_t.x));

    accLaunchKernel((add_pw_ekin_gpu_kernel<double>), dim3(grid_b), dim3(grid_t), 0, 0, num_gvec__, alpha__, pw_ekin__,
                    phi__, vphi__, hphi__);
}
